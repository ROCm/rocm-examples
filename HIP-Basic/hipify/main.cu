// MIT License
//
// Copyright (c) 2015-2022 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <hip/hip_runtime.h>

#include <cstdlib>
#include <iostream>
#include <vector>

#define CHECK(cmd)                                                                          \
    {                                                                                       \
        hipError_t error = cmd;                                                            \
        if(error != hipSuccess)                                                            \
        {                                                                                   \
            std::cerr << "error: " << hipGetErrorString(error) << " (" << error << ") at " \
                      << __FILE__ << ":" << __LINE__ << "\n";                               \
            exit(EXIT_FAILURE);                                                             \
        }                                                                                   \
    }

/// \brief Device function to square each element
/// in the array `in` and write to array `out`.
template<typename T>
__global__ void vector_square_kernel(T* out, T* in, const size_t size)
{
    // Get the unique global thread ID
    const size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    // Each thread hops stride amount of elements to find the next
    // element to square
    const size_t stride = blockDim.x * gridDim.x;

    for(size_t i = offset; i < size; i += stride)
    {
        out[i] = in[i] * in[i];
    }
}

int main()
{
    // Set the problem size
    constexpr size_t size          = 1000000;
    constexpr size_t size_in_bytes = size * sizeof(float);

    hipDeviceProp_t props;
    CHECK(hipGetDeviceProperties(&props, 0 /*deviceID*/));
    std::cout << "info: running on device " << props.name << "\n";

    std::cout << "info: allocate host mem (" << 2 * size_in_bytes / 1024.0 / 1024.0 << " MB) "
              << "\n";

    // Declare the host side arrays
    std::vector<float> h_in(size);
    std::vector<float> h_out(size);

    // Initialize the host size input
    for(size_t i = 0; i < size; i++)
    {
        h_in[i] = 1.618f + i;
    }
    // Declare the device side arrays
    float *d_in, *d_out;
    std::cout << "info: allocate device mem (" << 2 * size_in_bytes / 1024.0 / 1024.0 << " MB) "
              << "\n";
    // Allocate the device side memory
    CHECK(hipMalloc(&d_in, size_in_bytes));
    CHECK(hipMalloc(&d_out, size_in_bytes));

    std::cout << "info: copy Host2Device"
              << "\n";
    // Copy the input from host to the GPU device
    CHECK(hipMemcpy(d_in, h_in.data(), size_in_bytes, hipMemcpyHostToDevice));

    // Set the the number of blocks per kernel grid.
    constexpr unsigned int grid_size = 512;
    // Set the number of threads per kernel block.
    constexpr unsigned int threads_per_block = 256;

    std::cout << "info: launch 'vector_square_kernel' kernel"
              << "\n";
    vector_square_kernel<<<grid_size, threads_per_block>>>(d_out, d_in, size);

    std::cout << "info: copy Device2Host\n";
    CHECK(hipMemcpy(h_out.data(), d_out, size_in_bytes, hipMemcpyDeviceToHost));

    std::cout << "info: check result\n";
    for(size_t i = 0; i < size; i++)
    {
        if(h_out[i] != h_in[i] * h_in[i])
        {
            std::cerr << "FAILED! h_out[" << i << "] = " << h_out[i]
                      << ", expected:  " << h_in[i] * h_in[i] << '\n';
            exit(EXIT_FAILURE);
        }
    }
    std::cout << "PASSED!\n";
}
